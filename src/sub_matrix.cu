#include "sub_matrix.hpp"
#include "util.cuh"

#include <cassert>

namespace cudasw4{

    #ifdef __HIPCC__
    __constant__ char deviceSubMatrix[25*25];
    __constant__ int deviceSubMatrixDim;
    __constant__ int deviceSubMatrixDimSquared;
    #endif
    
    char hostSubMatrix[25*25];
    int hostSubMatrixDim;
    int hostSubMatrixDimSquared;
    
    //set host and device global variables
    
    
    void setProgramWideSubMatrix(SubMatrixType subMatrixType, const std::vector<int>& deviceIds){
        switch(subMatrixType){
            case SubMatrixType::DNA:
                {
                    const auto subMatrix = DNA::get1D();
                    const int dim = DNA::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);
                }
                break;               
            case SubMatrixType::NUC44:
                {
                    const auto subMatrix = NUC44::get1D();
                    const int dim = NUC44::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;            
            case SubMatrixType::PAM30:
                {
                    const auto subMatrix = PAM30::get1D();
                    const int dim = PAM30::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;            
            case SubMatrixType::PAM70:
                {
                    const auto subMatrix = PAM70::get1D();
                    const int dim = PAM70::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;            
            case SubMatrixType::BLOSUM45:
                {
                    const auto subMatrix = BLOSUM45::get1D();
                    const int dim = BLOSUM45::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM50:
                {
                    const auto subMatrix = BLOSUM50::get1D();
                    const int dim = BLOSUM50::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM62:
                {
                    const auto subMatrix = BLOSUM62::get1D();
                    const int dim = BLOSUM62::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM80:
                {
                    const auto subMatrix = BLOSUM80::get1D();
                    const int dim = BLOSUM80::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::PAM30_20:
                {
                    const auto subMatrix = PAM30_20::get1D();
                    const int dim = PAM30_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;                
            case SubMatrixType::PAM70_20:
                {
                    const auto subMatrix = PAM70_20::get1D();
                    const int dim = PAM70_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;                
            case SubMatrixType::BLOSUM45_20:
                {
                    const auto subMatrix = BLOSUM45_20::get1D();
                    const int dim = BLOSUM45_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM50_20:
                {
                    const auto subMatrix = BLOSUM50_20::get1D();
                    const int dim = BLOSUM50_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM62_20:
                {
                    const auto subMatrix = BLOSUM62_20::get1D();
                    const int dim = BLOSUM62_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            case SubMatrixType::BLOSUM80_20:
                {
                    const auto subMatrix = BLOSUM80_20::get1D();
                    const int dim = BLOSUM80_20::dim;
                    hostSubMatrixDim = dim;
                    hostSubMatrixDimSquared = dim * dim;
                    auto it = std::copy(subMatrix.begin(), subMatrix.end(), hostSubMatrix);
                    assert(std::distance(hostSubMatrix, it) <= 25 * 25);                
                }
                break;
            default:
                assert(false && "unimplemented substitution matrix copy");
                break;
        }
    #ifdef __HIPCC__
        RevertDeviceId rdi{};
    
        int numGpus = deviceIds.size();
    
        for(int gpu = 0; gpu < numGpus; gpu++){
            hipSetDevice(deviceIds[gpu]); CUERR;
            hipMemcpyToSymbol(HIP_SYMBOL(deviceSubMatrix), &(hostSubMatrix[0]), sizeof(char) * hostSubMatrixDim * hostSubMatrixDim); CUERR;
            hipMemcpyToSymbol(HIP_SYMBOL(deviceSubMatrixDim), &hostSubMatrixDim, sizeof(int)); CUERR;
            hipMemcpyToSymbol(HIP_SYMBOL(deviceSubMatrixDimSquared), &hostSubMatrixDimSquared, sizeof(int)); CUERR;
        }
    #endif    
    }

} //namespace cudasw4