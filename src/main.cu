#include <algorithm>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <unordered_set>

#include "hpc_helpers/all_helpers.cuh"
#include "hpc_helpers/peer_access.cuh"

// #include "kseqpp/kseqpp.hpp"
// #include "sequence_io.h"
#include "types.hpp"
#include "options.hpp"
#include "dbdata.hpp"
#include "cudasw4.cuh"
#include "config.hpp"
#include "reverse.hpp"
#include "alignments.hpp"

int main(int argc, char* argv[])
{
    ProgramOptions options;
    bool parseSuccess = parseArgs(argc, argv, options);

    if (options.version) {
        printVersion();
        return 0;
    }

    if(!parseSuccess || options.help) {
        printHelp(argc, argv);
        return 0;
    }

    if (options.reverseComplement) {
        if(options.sequenceType == cudasw4::SequenceType::Nucleotide) {
            const size_t numQueries = options.queries.size();
            for (size_t i = 0; i < numQueries; i++) {
                std::string reverseComplement = getReverseComplement(options.queries[i].sequence);
                options.queries.push_back(cudasw4::QuerySequence(options.queries[i].header + "_reversed", reverseComplement));
            }
        } else {
            options.reverseComplement = false;
        }
    }

    printOptions(options);

    std::vector<int> deviceIds;
    {
        int num = 0;
        hipGetDeviceCount(&num); CUERR
        for(int i = 0; i < num; i++){
            deviceIds.push_back(i);
        }
        if(deviceIds.size() > 0){
            if(options.verbose){
                std::cout << "Will use GPU";
                for(auto x : deviceIds){
                    std::cout << " " << x;
                }
                std::cout << "\n";
            }
        }else{
            throw std::runtime_error("No GPU found");
        }
    }

    helpers::PeerAccess peerAccess(deviceIds, false);
 
    using KernelTypeConfig = cudasw4::KernelTypeConfig;
    using MemoryConfig = cudasw4::MemoryConfig;
    using ScanResult = cudasw4::ScanResult;

    KernelTypeConfig kernelTypeConfig;
    kernelTypeConfig.singlePassType = options.singlePassType;
    kernelTypeConfig.manyPassType_small = options.manyPassType_small;
    kernelTypeConfig.manyPassType_large = options.manyPassType_large;
    kernelTypeConfig.overflowType = options.overflowType;

    MemoryConfig memoryConfig;
    memoryConfig.maxBatchBytes = options.maxBatchBytes;
    memoryConfig.maxBatchSequences = options.maxBatchSequences;
    memoryConfig.maxTempBytes = options.maxTempBytes;
    memoryConfig.maxGpuMem = options.maxGpuMem;

    std::ofstream outputfile(options.outputfile);
    if(!bool(outputfile)){
        throw std::runtime_error("Cannot open file " + options.outputfile);
    }

    int progressFileDescriptor = options.progressPipePath.length() == 0 ? -1 : open(options.progressPipePath.c_str(), O_WRONLY | O_NONBLOCK);

    cudasw4::CudaSW4 cudaSW4(
        deviceIds, 
        options.numTopOutputs,
        options.sequenceType,
        options.subMatrixType, 
        kernelTypeConfig, 
        memoryConfig, 
        options.verbose,
        options.progressKey,
        progressFileDescriptor
    );

    size_t db_count = options.databases.size();

    std::unordered_set<std::string> uniqueHeaders;
    std::vector<cudasw4::HitResult> totalResults;

    for (size_t db_index = 0; db_index < db_count; db_index ++) {

        if(options.verbose) {
            std::cout << "Reading Database: \n";
        }
        try {
            helpers::CpuTimer timer_read_db("Read DB");
            constexpr bool writeAccess = false;
            const bool prefetchSeq = options.prefetchDBFile;

            auto fullDB_tmp = std::make_shared<cudasw4::DB>(cudasw4::loadDB(options.databases[db_index], writeAccess, prefetchSeq));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        } catch(cudasw4::LoadDBException& ex) {
            if(options.verbose){
                std::cout << "Failed to map db files. Using fallback db. Error message: " << ex.what() << "\n";
            }
            helpers::CpuTimer timer_read_db("Read DB");
            auto fullDB_tmp = std::make_shared<cudasw4::DBWithVectors>(cudasw4::loadDBWithVectors(options.databases[db_index]));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        }

        if (options.verbose) {
            cudaSW4.printDBInfo();
            if(options.printLengthPartitions){
                cudaSW4.printDBLengthPartitions();
            }
        }

        if (options.loadFullDBToGpu) {
            cudaSW4.prefetchFullDBToGpus();
        }

        size_t queries_count = options.queries.size();
        size_t query_num = 0;
        for (const auto& query : options.queries) {

            cudaSW4.totalTimerStart();

            cudasw4::DbQueryIndex scan_index(db_index, db_count, query_num, queries_count);

            std::cout << "Processing query " << query_num << " ... ";
            std::cout.flush();
            const std::string& header   = query.header;
            const std::string& sequence = query.sequence;

            ScanResult scanResult = cudaSW4.scan(scan_index, sequence.data(), sequence.size());

            if(options.verbose){
                std::cout << "Done. Scan time: " << scanResult.stats.seconds << " s, " << scanResult.stats.gcups << " GCUPS\n";
            }else{
                std::cout << "Done.\n";
            }

            for (size_t i = 0; i < scanResult.scores.size(); ++i) {
                if (scanResult.scores[i] < options.minScore) break;

                std::string refHeader = (std::string)cudaSW4.getReferenceHeader(scanResult.referenceIds[i]);

                if (uniqueHeaders.find(refHeader) == uniqueHeaders.end()) {
                    // The header is not in the set, so it's a new unique element
                    // std::string refSequence = (std::string)cudaSW4.getReferenceSequence(scanResult.referenceIds[i]);
                    totalResults.push_back(
                        cudasw4::HitResult(
                            query_num, 
                            header,
                            sequence.size(),
                            sequence,
                            refHeader,
                            cudaSW4.getReferenceLength(scanResult.referenceIds[i]),
                            //TODO: think about how to store subject sequence for final alignment calculation
                            cudaSW4.getReferenceSequence(scanResult.referenceIds[i]), 
                            scanResult.scores[i]
                        )
                    );
                    uniqueHeaders.insert(refHeader); // Add the header to the set
                }
            }

            query_num++;

            auto totalBenchmarkStats = cudaSW4.totalTimerStop();
            if(options.verbose){
                std::cout << "Total time: " << totalBenchmarkStats.seconds << " s, " << totalBenchmarkStats.gcups << " GCUPS\n";
            }
        }
    }

    if(options.numTopOutputs > 0){
        std::sort(totalResults.begin(), totalResults.end(), [](const auto& a, const auto& b) {
            return a.score > b.score;
        });

        cudasw2::calculateAlignments(
            outputfile, 
            totalResults, 
            options,
            cudasw4::hostSubMatrix, 
            cudasw4::hostSubMatrixDim
        );

        // cudasw2::printScanResultCSV(outputfile, totalResults, options);
        
        outputfile.flush();
    }

    if (progressFileDescriptor != -1) close(progressFileDescriptor);
}
