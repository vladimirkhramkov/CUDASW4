#include <algorithm>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "hpc_helpers/all_helpers.cuh"
#include "hpc_helpers/peer_access.cuh"

#include "kseqpp/kseqpp.hpp"
#include "sequence_io.h"
#include "options.hpp"
#include "dbdata.hpp"
#include "cudasw4.cuh"
#include "config.hpp"


std::vector<std::string> split(const std::string& str, char c){
	std::vector<std::string> result;

	std::stringstream ss(str);
	std::string s;

	while (std::getline(ss, s, c)) {
		result.emplace_back(s);
	}

	return result;
}

void printScanResultPlain(std::ostream& os, const cudasw4::ScanResult& scanResult, const cudasw4::CudaSW4& cudaSW4){
    const int n = scanResult.scores.size();
    for(int i = 0; i < n; i++){
        const auto referenceId = scanResult.referenceIds[i];
        os << "Result " << i << ".";
        os << " Score: " << scanResult.scores[i] << ".";
        os << " Length: " << cudaSW4.getReferenceLength(referenceId) << ".";
        os << " Header " << cudaSW4.getReferenceHeader(referenceId) << ".";
        os << " referenceId " << referenceId;
        os << "\n";
        //std::cout << " Sequence " << cudaSW4.getReferenceSequence(referenceId) << "\n";

    }
}

void printTSVHeader(std::ostream& os){
    constexpr char sep = '\t';

    os << "Query number" << sep 
        << "Query length" << sep 
        << "Query header" << sep
        << "Result number" << sep
        << "Result score" << sep
        << "Reference length" << sep
        << "Reference header" << sep
        << "Reference ID in DB" << "\n";
}

void printScanResultTSV(
    std::ostream& os, 
    const cudasw4::ScanResult& scanResult, 
    const cudasw4::CudaSW4& cudaSW4, 
    int64_t queryId,
    cudasw4::SequenceLengthT queryLength,
    std::string_view queryHeader
){
    constexpr char sep = '\t';

    const int n = scanResult.scores.size();
    for(int i = 0; i < n; i++){
        const auto referenceId = scanResult.referenceIds[i];
        
        os << queryId << sep 
            << queryLength << sep
            << queryHeader << sep
            << i << sep
            << scanResult.scores[i] << sep
            << cudaSW4.getReferenceLength(referenceId) << sep
            << cudaSW4.getReferenceHeader(referenceId) << sep
            << referenceId << "\n";

        //std::cout << " Sequence " << cudaSW4.getReferenceSequence(referenceId) << "\n";
    }
}

struct BatchOfQueries{
    std::vector<char> chars;               
    std::vector<std::size_t> offsets;  
    std::vector<cudasw4::SequenceLengthT> lengths;  
    std::vector<std::string> headers;  
};

int main(int argc, char* argv[])
{
    ProgramOptions options;
    bool parseSuccess = parseArgs(argc, argv, options);

    if (options.version) {
        printVersion();
        return 0;
    }

    if(!parseSuccess || options.help){
        printHelp(argc, argv);
        return 0;
    }

    printOptions(options);

    std::vector<int> deviceIds;
    {
        int num = 0;
        hipGetDeviceCount(&num); CUERR
        for(int i = 0; i < num; i++){
            deviceIds.push_back(i);
        }
        if(deviceIds.size() > 0){
            if(options.verbose){
                std::cout << "Will use GPU";
                for(auto x : deviceIds){
                    std::cout << " " << x;
                }
                std::cout << "\n";
            }
        }else{
            throw std::runtime_error("No GPU found");
        }
    }

    helpers::PeerAccess peerAccess(deviceIds, false);
 
    using KernelTypeConfig = cudasw4::KernelTypeConfig;
    using MemoryConfig = cudasw4::MemoryConfig;
    using ScanResult = cudasw4::ScanResult;

    KernelTypeConfig kernelTypeConfig;
    kernelTypeConfig.singlePassType = options.singlePassType;
    kernelTypeConfig.manyPassType_small = options.manyPassType_small;
    kernelTypeConfig.manyPassType_large = options.manyPassType_large;
    kernelTypeConfig.overflowType = options.overflowType;

    MemoryConfig memoryConfig;
    memoryConfig.maxBatchBytes = options.maxBatchBytes;
    memoryConfig.maxBatchSequences = options.maxBatchSequences;
    memoryConfig.maxTempBytes = options.maxTempBytes;
    memoryConfig.maxGpuMem = options.maxGpuMem;

    std::ofstream outputfile(options.outputfile);
    if(!bool(outputfile)){
        throw std::runtime_error("Cannot open file " + options.outputfile);
    }

    if(options.outputMode == ProgramOptions::OutputMode::TSV){
        printTSVHeader(outputfile);
    }

    int progressFileDescriptor = options.progressPipePath.length() == 0 ? -1 : open(options.progressPipePath.c_str(), O_WRONLY | O_NONBLOCK);

    cudasw4::CudaSW4 cudaSW4(
        deviceIds, 
        options.numTopOutputs,
        options.sequenceType,
        options.subMatrixType, 
        kernelTypeConfig, 
        memoryConfig, 
        options.verbose,
        options.progressKey,
        progressFileDescriptor
    );

    if(!options.usePseudoDB){
        if(options.verbose){
            std::cout << "Reading Database: \n";
        }
        try{
            helpers::CpuTimer timer_read_db("Read DB");
            constexpr bool writeAccess = false;
            const bool prefetchSeq = options.prefetchDBFile;

            auto fullDB_tmp = std::make_shared<cudasw4::DB>(cudasw4::loadDB(options.dbPrefix, writeAccess, prefetchSeq));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        }catch(cudasw4::LoadDBException& ex){
            if(options.verbose){
                std::cout << "Failed to map db files. Using fallback db. Error message: " << ex.what() << "\n";
            }
            helpers::CpuTimer timer_read_db("Read DB");
            auto fullDB_tmp = std::make_shared<cudasw4::DBWithVectors>(cudasw4::loadDBWithVectors(options.dbPrefix));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        }
    }

    if(options.verbose){
        cudaSW4.printDBInfo();
        if(options.printLengthPartitions){
            cudaSW4.printDBLengthPartitions();
        }
    }

    if(options.loadFullDBToGpu){
        cudaSW4.prefetchFullDBToGpus();
    }

    //non interactive mode
    for(const auto& queryFile : options.queryFiles){
        std::cout << "Processing query file " << queryFile << "\n";

        kseqpp::KseqPP reader(queryFile);
        int64_t query_num = 0;

        cudaSW4.totalTimerStart();

        while(reader.next() >= 0){
            std::cout << "Processing query " << query_num << " ... ";
            std::cout.flush();
            const std::string& header = reader.getCurrentHeader();
            const std::string& sequence = reader.getCurrentSequence();

            ScanResult scanResult = cudaSW4.scan(sequence.data(), sequence.size());
            if(options.verbose){
                std::cout << "Done. Scan time: " << scanResult.stats.seconds << " s, " << scanResult.stats.gcups << " GCUPS\n";
            }else{
                std::cout << "Done.\n";
            }

            if(options.numTopOutputs > 0){
                if(options.outputMode == ProgramOptions::OutputMode::Plain){
                    outputfile << "Query " << query_num << ", header" <<  header
                        << ", length " << sequence.size()
                        << ", num overflows " << scanResult.stats.numOverflows << "\n";

                    printScanResultPlain(outputfile, scanResult, cudaSW4);
                }else{
                    printScanResultTSV(outputfile, scanResult, cudaSW4, query_num, sequence.size(), header);
                }
                outputfile.flush();
            }

            query_num++;
        }

        auto totalBenchmarkStats = cudaSW4.totalTimerStop();
        if(options.verbose){
            std::cout << "Total time: " << totalBenchmarkStats.seconds << " s, " << totalBenchmarkStats.gcups << " GCUPS\n";
        }
    }

    if (progressFileDescriptor != -1) close(progressFileDescriptor);
}
