#include <algorithm>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>

#include "hpc_helpers/all_helpers.cuh"
#include "hpc_helpers/peer_access.cuh"

// #include "kseqpp/kseqpp.hpp"
// #include "sequence_io.h"
#include "options.hpp"
#include "dbdata.hpp"
#include "cudasw4.cuh"
#include "config.hpp"
#include "reverse.hpp"

std::vector<std::string> split(const std::string& str, char c){
	std::vector<std::string> result;

	std::stringstream ss(str);
	std::string s;

	while (std::getline(ss, s, c)) {
		result.emplace_back(s);
	}

	return result;
}

void printScanResultPlain(
    std::ostream& os, 
    const cudasw4::ScanResult& scanResult, 
    const cudasw4::CudaSW4& cudaSW4, 
    const ProgramOptions& options
){
    const int n = scanResult.scores.size();
    for(int i = 0; i < n; i++){
        if (scanResult.scores[i] < options.minScore) continue;

        const auto referenceId = scanResult.referenceIds[i];
        os << "Result " << i << ".";
        os << " Score: " << scanResult.scores[i] << ".";
        os << " Length: " << cudaSW4.getReferenceLength(referenceId) << ".";
        os << " Header " << cudaSW4.getReferenceHeader(referenceId) << ".";
        os << " referenceId " << referenceId;
        os << "\n";
        //std::cout << " Sequence " << cudaSW4.getReferenceSequence(referenceId) << "\n";

    }
}

void printScanResultCSV(
    std::ostream& os, 
    const cudasw4::ScanResult& scanResult, 
    const cudasw4::CudaSW4& cudaSW4, 
    const ProgramOptions& options,
    int64_t queryId,
    cudasw4::SequenceLengthT queryLength,
    std::string_view queryHeader
){
    constexpr char sep = ',';

    const int n = scanResult.scores.size();
    for(int i = 0; i < n; i++){
        if (scanResult.scores[i] < options.minScore) continue;

        const auto referenceId = scanResult.referenceIds[i];
        
        // os << queryId << sep 
        //     << queryLength << sep
        //     << queryHeader << sep
        //     << i << sep
        //     << scanResult.scores[i] << sep
        //     << cudaSW4.getReferenceLength(referenceId) << sep
        //     << cudaSW4.getReferenceHeader(referenceId) << sep
        //     << referenceId << "\n";

        //std::cout << " Sequence " << cudaSW4.getReferenceSequence(referenceId) << "\n";

        for (unsigned int paramIndex = 0; paramIndex < options.csvColumns.size(); ++paramIndex) {
            std::string parameter = options.csvColumns[paramIndex];
            if (paramIndex)
                os << sep;

            if (parameter == "qacc") {
                os << queryHeader || queryId; //initialQueries[answerEntry.queryIndex].first;
            } else if (parameter == "qlen") {
                os << queryLength; //initialQueries[answerEntry.queryIndex].second.length();
            } else if (parameter == "sacc") {
                os << cudaSW4.getReferenceHeader(referenceId);//answerEntry.getSacc();
            } else if (parameter == "slen") {
                os << cudaSW4.getReferenceLength(referenceId); //answerEntry.getSlen();
            } else if (parameter == "score") {
                os << scanResult.scores[i]; //answerEntry.getScore();
            } else if (parameter == "length") {
                // os << stringAlignment[0].length();
            } else if (parameter == "nident") {
                // int matches = 0;
                // for (int i = 0; i < stringAlignment[0].length(); ++i)
                //     if (stringAlignment[0][i] != '-' && (stringAlignment[0][i] == stringAlignment[2][i] || stringAlignment[1][i] == '|' && nucleotideSearch))
                //         ++matches;
                // os << matches;
            } else if (parameter == "gaps") {
                // int gaps = 0;
                // for (int i = 0; i < stringAlignment[0].length(); ++i)
                //     gaps += (stringAlignment[0][i] == '-')
                //             + (stringAlignment[2][i] == '-');
                // os << gaps;
            } else if (parameter == "qstart") {
                // os << answerEntry.getQstart() + 1;
            } else if (parameter == "qend") {
                // os << answerEntry.getQend() + 1;
            } else if (parameter == "sstart") {
                // os << answerEntry.getSstart() + 1;
            } else if (parameter == "send") {
                // os << answerEntry.getSend() + 1;
            } else if (parameter == "positive") {
                // int positiveMatches = 0;
                // for (int i = 0; i < stringAlignment[0].length(); ++i) {
                //     if (stringAlignment[0][i] != '-'
                //             && stringAlignment[2][i] != '-') {
                //         positiveMatches +=
                //                 scoringMatrix[CFastaFile::amino_acids_trans[stringAlignment[0][i]]][CFastaFile::amino_acids_trans[stringAlignment[2][i]]]
                //                         > 0;
                //     }
                // }
                // os << positiveMatches;
            } else if (parameter == "btop") {
                // os << BTOP;
            } else if (parameter == "topline") {
                // os << stringAlignment[0];
            } else if (parameter == "middleline") {
                // os << stringAlignment[1];
            } else if (parameter == "bottomline") {
                // os << stringAlignment[2];
            } else if (parameter == "reversed") {
                // os << answerEntry.reversed;
            } else if (parameter == "qcovs") {
                // int coverageLength = queryCoverageLength[answerEntry.sequenceName];
                // os << fixed << setprecision(2) << 100.0 * coverageLength / query.length();
                // rounded percent
                //os << (200 * coverageLength + query.length()) / (2 * query.length());
            }
        }
        os << "\n";

    }
}

struct BatchOfQueries{
    std::vector<char> chars;               
    std::vector<std::size_t> offsets;  
    std::vector<cudasw4::SequenceLengthT> lengths;  
    std::vector<std::string> headers;  
};

int main(int argc, char* argv[])
{
    ProgramOptions options;
    bool parseSuccess = parseArgs(argc, argv, options);

    if (options.version) {
        printVersion();
        return 0;
    }

    if(!parseSuccess || options.help) {
        printHelp(argc, argv);
        return 0;
    }

    if (options.reverseComplement) {
        if(options.sequenceType == cudasw4::SequenceType::Nucleotide) {
            const size_t numQueries = options.queries.size();
            for (size_t i = 0; i < numQueries; i++) {
                std::string reverseComplement = getReverseComplement(options.queries[i].sequence);
                options.queries.push_back(cudasw4::QuerySequence(options.queries[i].header + "_reversed", reverseComplement));
            }
        } else {
            options.reverseComplement = false;
        }
    }

    printOptions(options);

    std::vector<int> deviceIds;
    {
        int num = 0;
        hipGetDeviceCount(&num); CUERR
        for(int i = 0; i < num; i++){
            deviceIds.push_back(i);
        }
        if(deviceIds.size() > 0){
            if(options.verbose){
                std::cout << "Will use GPU";
                for(auto x : deviceIds){
                    std::cout << " " << x;
                }
                std::cout << "\n";
            }
        }else{
            throw std::runtime_error("No GPU found");
        }
    }

    helpers::PeerAccess peerAccess(deviceIds, false);
 
    using KernelTypeConfig = cudasw4::KernelTypeConfig;
    using MemoryConfig = cudasw4::MemoryConfig;
    using ScanResult = cudasw4::ScanResult;

    KernelTypeConfig kernelTypeConfig;
    kernelTypeConfig.singlePassType = options.singlePassType;
    kernelTypeConfig.manyPassType_small = options.manyPassType_small;
    kernelTypeConfig.manyPassType_large = options.manyPassType_large;
    kernelTypeConfig.overflowType = options.overflowType;

    MemoryConfig memoryConfig;
    memoryConfig.maxBatchBytes = options.maxBatchBytes;
    memoryConfig.maxBatchSequences = options.maxBatchSequences;
    memoryConfig.maxTempBytes = options.maxTempBytes;
    memoryConfig.maxGpuMem = options.maxGpuMem;

    std::ofstream outputfile(options.outputfile);
    if(!bool(outputfile)){
        throw std::runtime_error("Cannot open file " + options.outputfile);
    }

    int progressFileDescriptor = options.progressPipePath.length() == 0 ? -1 : open(options.progressPipePath.c_str(), O_WRONLY | O_NONBLOCK);

    cudasw4::CudaSW4 cudaSW4(
        deviceIds, 
        options.numTopOutputs,
        options.sequenceType,
        options.subMatrixType, 
        kernelTypeConfig, 
        memoryConfig, 
        options.verbose,
        options.progressKey,
        progressFileDescriptor
    );

    if(!options.usePseudoDB){
        if(options.verbose){
            std::cout << "Reading Database: \n";
        }
        try{
            helpers::CpuTimer timer_read_db("Read DB");
            constexpr bool writeAccess = false;
            const bool prefetchSeq = options.prefetchDBFile;

            auto fullDB_tmp = std::make_shared<cudasw4::DB>(cudasw4::loadDB(options.dbPrefix, writeAccess, prefetchSeq));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        }catch(cudasw4::LoadDBException& ex){
            if(options.verbose){
                std::cout << "Failed to map db files. Using fallback db. Error message: " << ex.what() << "\n";
            }
            helpers::CpuTimer timer_read_db("Read DB");
            auto fullDB_tmp = std::make_shared<cudasw4::DBWithVectors>(cudasw4::loadDBWithVectors(options.dbPrefix));
            if(options.verbose){
                timer_read_db.print();
            }

            cudaSW4.setDatabase(fullDB_tmp);
        }
    }

    if(options.verbose){
        cudaSW4.printDBInfo();
        if(options.printLengthPartitions){
            cudaSW4.printDBLengthPartitions();
        }
    }

    if(options.loadFullDBToGpu){
        cudaSW4.prefetchFullDBToGpus();
    }

    int64_t query_num = 0;
    for(const auto& query : options.queries){

        cudaSW4.totalTimerStart();

        std::cout << "Processing query " << query_num << " ... ";
        std::cout.flush();
        const std::string& header   = query.header;
        const std::string& sequence = query.sequence;

        ScanResult scanResult = cudaSW4.scan(sequence.data(), sequence.size());
        if(options.verbose){
            std::cout << "Done. Scan time: " << scanResult.stats.seconds << " s, " << scanResult.stats.gcups << " GCUPS\n";
        }else{
            std::cout << "Done.\n";
        }

        if(options.numTopOutputs > 0){
            if(options.outputMode == ProgramOptions::OutputMode::Plain){
                printScanResultPlain(outputfile, scanResult, cudaSW4, options);
            }else{
                printScanResultCSV(outputfile, scanResult, cudaSW4, options, query_num, sequence.size(), header);
            }
            outputfile.flush();
        }

        query_num++;

        auto totalBenchmarkStats = cudaSW4.totalTimerStop();
        if(options.verbose){
            std::cout << "Total time: " << totalBenchmarkStats.seconds << " s, " << totalBenchmarkStats.gcups << " GCUPS\n";
        }
    }

    if (progressFileDescriptor != -1) close(progressFileDescriptor);
}
